
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void convertToGray(int pixel, unsigned char *red, unsigned char *green, unsigned char *blue, unsigned char *out)
{
    //int i = blockIdx.x*blockDim.x + threadIdx.x;
    for (int j = blockIdx.x; j < 100000; j+=gridDim.x)
    {
        for (int i = threadIdx.x; i < pixel; i+=blockDim.x)
        {
        out[i] = (red[i] + green[i] + blue[i]) / 3;
        /*printf("%d,", out[i]);*/
        }
    }
}

int main()
{
    FILE *fp;
    int width, height, pixel, i;
    unsigned char *red, *green, *blue, *out;
    unsigned char *d_red, *d_green, *d_blue, *d_out;



    fp = fopen("image.txt", "r");

    fscanf(fp, "%d %d %d", &width, &height, &pixel);

    // Allocate host memory
    red   = (unsigned char *)malloc(pixel * sizeof(unsigned char));
    green = (unsigned char *)malloc(pixel * sizeof(unsigned char));
    blue  = (unsigned char *)malloc(pixel * sizeof(unsigned char));
    out   = (unsigned char *)malloc(pixel * sizeof(unsigned char));

    for ( i = 0; i < pixel; i++) fscanf(fp, "%hhu %hhu %hhu", &red[i], &green[i], &blue[i]);

    fclose(fp);


    clock_t start = clock();

    // Allocate device memory
    hipMalloc((void**)&d_red,   pixel * sizeof(unsigned char));
    hipMalloc((void**)&d_green, pixel * sizeof(unsigned char));
    hipMalloc((void**)&d_blue,  pixel * sizeof(unsigned char));
    hipMalloc((void**)&d_out,   pixel * sizeof(unsigned char));

    // Transfer data from host to device memory
    hipMemcpy(d_red,   red,   pixel * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_green, green, pixel * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_blue,  blue,  pixel * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_out,   out,   pixel * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Main function
    convertToGray<<< 1024, 1024>>>(pixel, d_red, d_green, d_blue, d_out);

    // Transfer data from device to host memory
    hipMemcpy(out, d_out, pixel * sizeof(unsigned char), hipMemcpyDeviceToHost);

    clock_t end = clock();

    // printf("\nwidth = %d\nheight = %d\npixel = %d\n", width, height, pixel);

    printf( "clock:%f\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Deallocate device memory
    hipFree(d_red);
    hipFree(d_green);
    hipFree(d_blue);
    hipFree(d_out);

    // Deallocate host memory
    free(red);
    free(green);
    free(blue);
    free(out);

    return 0;
}