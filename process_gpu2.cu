
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void convertToGray(int pixel, int cycle, unsigned char *red, unsigned char *green, unsigned char *blue, unsigned char *out)
{
    for (int j = blockIdx.x; j < cycle; j+=gridDim.x)
    {
        for (int i = threadIdx.x; i < pixel; i+=blockDim.x)
        {
        out[i] = (red[i] + green[i] + blue[i]) / 3;
        /*printf("%d,", out[i]);*/
        }
    }
}

int main()
{
    int width, height, pixel, cycle, i;
    unsigned char *red,   *green,   *blue,   *out;
    unsigned char *d_red, *d_green, *d_blue, *d_out;

    /* input */
    printf("width?  >>> ");
    scanf("%d", &width);
    printf("height? >>> ");
    scanf("%d", &height);
    printf("cycle?  >>> ");
    scanf("%d", &cycle);

    pixel = width * height;

    /* Allocate host memory */
    red   = (unsigned char *)malloc(pixel * sizeof(unsigned char));
    green = (unsigned char *)malloc(pixel * sizeof(unsigned char));
    blue  = (unsigned char *)malloc(pixel * sizeof(unsigned char));
    out   = (unsigned char *)malloc(pixel * sizeof(unsigned char));

    /* Fail to assgin */
    if ( red == NULL || pixel > 100000000 || cycle > 1000000 )
    {
        printf("too many pixels or cycles...");
        return 0;
    }

    /* Make value of RGB */
    for ( i = 0; i < pixel; i++) red[i]   = rand() % 255;
    for ( i = 0; i < pixel; i++) green[i] = rand() % 255;
    for ( i = 0; i < pixel; i++) blue[i]  = rand() % 255;

    /* Start timer */
    clock_t start = clock();

    /* Allocate device memory */
    hipMalloc((void**)&d_red,   pixel * sizeof(unsigned char));
    hipMalloc((void**)&d_green, pixel * sizeof(unsigned char));
    hipMalloc((void**)&d_blue,  pixel * sizeof(unsigned char));
    hipMalloc((void**)&d_out,   pixel * sizeof(unsigned char));

    /* Transfer data from host to device memory */
    hipMemcpy(d_red,   red,   pixel * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_green, green, pixel * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_blue,  blue,  pixel * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_out,   out,   pixel * sizeof(unsigned char), hipMemcpyHostToDevice);

    /* Main function */
    convertToGray<<< 512,1024>>>(pixel, cycle, d_red, d_green, d_blue, d_out);

    /* Transfer data from device to host memory */
    hipMemcpy(out, d_out, pixel * sizeof(unsigned char), hipMemcpyDeviceToHost);

    /* Stop timer */
    clock_t end = clock();

    /* Show result */
    printf("\n(width, height, pixel, cycle) = (%d, %d, %d, %d)\n", width, height, pixel, cycle);
    printf( "clock:%f\n", (double)(end - start) / CLOCKS_PER_SEC);

    /* Deallocate device memory */
    hipFree(d_red);
    hipFree(d_green);
    hipFree(d_blue);
    hipFree(d_out);

    /* Deallocate host memory */
    free(red);
    free(green);
    free(blue);
    free(out);

    return 0;
}